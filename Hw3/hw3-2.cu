#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define INF (1 << 30) - 1
#define B 32

//======================
#define DEV_NO 0
hipDeviceProp_t prop;

int *Dist;
int V, E, V_padding, rounds;

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&V, sizeof(int), 1, file);
    fread(&E, sizeof(int), 1, file);

    V_padding = (V % B == 0) ? V : (V / B + 1) * B;
    Dist = (int *)malloc(sizeof(int) * V_padding * V_padding);

    for (int i = 0; i < V_padding; ++i)
        for (int j = 0; j < V_padding; ++j)
            Dist[i * V_padding + j] = (i == j) ? 0 : INF;

    int pair[3];
    for (int i = 0; i < E; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * V_padding + pair[1]] = pair[2];
    }
    fclose(file);
    rounds = ceil(V_padding / B);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < V; ++i) {
        for (int j = 0; j < V; ++j)
            if (Dist[i * V_padding + j] >= INF) Dist[i * V_padding + j] = INF;
        fwrite(&Dist[i * V_padding], sizeof(int), V, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void phase1(int *d, int round, int v) {
    int i = threadIdx.y + round * B;
    int j = threadIdx.x + round * B;

    if (i < v && j < v) {
        __shared__ int d_shared[B * B];

        int sharedIndexY = threadIdx.y * B;

        d_shared[sharedIndexY + threadIdx.x] = d[i * v + j];
        __syncthreads();

        for (int k = 0; k < B; k++) {
            int newWeight = d_shared[sharedIndexY + k] + d_shared[k * B + threadIdx.x];
            if (newWeight < d_shared[sharedIndexY + threadIdx.x])
                d_shared[sharedIndexY + threadIdx.x] = newWeight;
            __syncthreads();
        }

        d[i * v + j] = d_shared[sharedIndexY + threadIdx.x];
    }
}

__global__ void phase2(int *d, int round, int v) {
    if (blockIdx.x == round) // pivot block
        return;

    int i = threadIdx.y + blockIdx.x * B;
    int j = threadIdx.x + blockIdx.x * B;
    int pivot_i = threadIdx.y + round * B;
    int pivot_j = threadIdx.x + round * B;

    if (blockIdx.y == 0) // pivot row
        i = pivot_i;
    else // pivot column
        j = pivot_j;

    if (i < v && j < v) {
        __shared__ int pivot_shared[B * B];
        __shared__ int d_shared[B * B];

        int sharedIndexY = threadIdx.y * B;

        pivot_shared[sharedIndexY + threadIdx.x] = d[pivot_i * v + pivot_j];
        d_shared[sharedIndexY + threadIdx.x] = d[i * v + j];
        __syncthreads();

        for (int k = 0; k < B; k++) {
            int newWeight;
            if (blockIdx.y == 0)
                newWeight = pivot_shared[sharedIndexY + k] + d_shared[k * B + threadIdx.x];
            else
                newWeight = d_shared[sharedIndexY + k] + pivot_shared[k * B + threadIdx.x];

            if (newWeight < d_shared[sharedIndexY + threadIdx.x])
                d_shared[sharedIndexY + threadIdx.x] = newWeight;
        }

        d[i * v + j] = d_shared[sharedIndexY + threadIdx.x];
    }
}

__global__ void phase3(int *d, int round, int v) {
    if (blockIdx.x == round || blockIdx.y == round) // calculated
        return;

    int i = threadIdx.y + blockIdx.y * B;
    int j = threadIdx.x + blockIdx.x * B;

    if (i < v && j < v) {
        __shared__ int row_shared[B * B];
        __shared__ int col_shared[B * B];

        int sharedIndexY = threadIdx.y * B;

        row_shared[sharedIndexY + threadIdx.x] = d[i * v + (threadIdx.x + round * B)];
        col_shared[sharedIndexY + threadIdx.x] = d[(threadIdx.y + round * B) * v + j];
        __syncthreads();

        int weight = d[i * v + j];
        for (int k = 0; k < B; k++) {
            int newWeight = row_shared[sharedIndexY + k] + col_shared[k * B + threadIdx.x];
            if (newWeight < weight)
                weight = newWeight;
        }

        d[i * v + j] = weight;
    }
}

void block_FW(void) {
    int *Dist_device;
    hipMalloc(&Dist_device, sizeof(int) * V_padding * V_padding);
    hipMemcpy(Dist_device, Dist, sizeof(int) * V_padding * V_padding, hipMemcpyHostToDevice);

    dim3 num_threads(B, B), num_blocks_2(rounds, 2), num_blocks_3(rounds, rounds);

    for (int round = 0; round < rounds; round++) {
        phase1<<<1, num_threads>>>(Dist_device, round, V_padding);
        phase2<<<num_blocks_2, num_threads>>>(Dist_device, round, V_padding);
        phase3<<<num_blocks_3, num_threads>>>(Dist_device, round, V_padding);
    }

    hipMemcpy(Dist, Dist_device, sizeof(int) * V_padding * V_padding, hipMemcpyDeviceToHost);
}

int main(int argc, char* argv[]) {
    hipGetDeviceProperties(&prop, DEV_NO);
    printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    input(argv[1]);
    printf("rounds: %d\n", rounds);
    block_FW();
    output(argv[2]);
    return 0;
}